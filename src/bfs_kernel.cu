#include "hip/hip_runtime.h"
#ifndef _BFS_KERNEL_H_
#define _BFS_KERNEL_H_

/******************************Include Files***********************************/
#include <iostream>
#include "Graph.h"
#include <stdio.h>
#include <fstream>
#include <thread>

/******************************Global Consts***********************************/
const int MAX_THREADS_PER_BLOCK = 512;



/****************************Function Prototypes*******************************/
void convertAdjListToArray(Graph * G, int * vertexArray, int * edgeArray);
void multipleGPUThreadFunction(int * vertexArray, int * edgeArray, int numVertices, int numEdges, int * source, int deviceNum, int ** gpuCost, int numIngress, int numDevices);



/******************************CUDA Kernels************************************/
__global__ void bfs_kernel(int * vertexArray, int * edgeArray, bool * frontierArray, bool * frontierUpdatingArray, bool * visitedArray, int * costArray, int numVertices )
{
	// get the thread id
    unsigned int tid = blockIdx.x * MAX_THREADS_PER_BLOCK + threadIdx.x;

    // check if the thread should process the vertex
    if(tid < numVertices && frontierArray[tid])
    {                   
    	// mark as the vertex has been processed
        frontierArray[tid] = false;

        // calulate the start and end index for the edge array
        int startIndex = vertexArray[tid];
        int endIndex = vertexArray[tid + 1];

        // loop through all the edges
        for(int i = startIndex; i < endIndex; i++)
        {                       
        	// get the destination vertex
            int destVertex = edgeArray[i];

            // if the destination has not been visited
            if(!visitedArray[destVertex])
            {
            	// increment the cost
                costArray[destVertex] = costArray[tid] + 1;
                
                // mark that the vertex needs to be processed
                frontierUpdatingArray[destVertex] = true;
            }
        }       
    }

}


__global__ void bfsUpdateArrays_kernel(bool * frontierArray, bool * frontierUpdatingArray, bool * visitedArray, bool * searching, int numVertices)
{
	// get the thread id
    unsigned int tid = blockIdx.x * MAX_THREADS_PER_BLOCK + threadIdx.x;

    // check if the thread should update the array for the vertex
    if(tid < numVertices && frontierUpdatingArray[tid])
    {
        frontierArray[tid] = true;
        visitedArray[tid] = true;
        *searching = true;
        frontierUpdatingArray[tid] = false;
    }
}



/*************************Function Implementation*******************************/
float processGraphSingleGPU(Graph * graph, int * source, int numIngress, int ** gpuCost)
    {
        // declare variables
        bool searching = true;
  		hipEvent_t start, end;

        // get the number of vertices and edges
        int numVertices = graph->numberOfVertex();
        int numEdges = graph->numberOfEdges() * 2;

		// create device event timers
		hipEventCreate(&start);
		hipEventCreate(&end);

        // allocate the memory for the arrays
        int * vertexArray = new int [numVertices + 1];
        int * edgeArray = new int [numEdges];

        // start GPU timer
        hipEventRecord( start, 0 );

        convertAdjListToArray(graph, vertexArray, edgeArray);


   
		multipleGPUThreadFunction(vertexArray, edgeArray, numVertices, numEdges, source, 0, gpuCost, numIngress, 1);
		

        // stop the GPU timer
        hipEventRecord( end, 0 );
        hipEventSynchronize( end );


        // calculate the amount of execution time
        float deviceWithTransferTime;
        hipEventElapsedTime( &deviceWithTransferTime, start, end );

        // clean up timer variables
        hipEventDestroy(start);
    	hipEventDestroy(end);   

        // clean up memory
        delete [] vertexArray;
        delete [] edgeArray;    		

        // return the execution time
   	    return (deviceWithTransferTime / 1000.0);
    }


float processGraphMultipleGPU(Graph * graph, int * source, int numIngress, int ** gpuCost, int numDevices)
    {
        // declare variables
  		hipEvent_t start, end;
  		int deviceThreads = 0;

        // get the number of vertices and edges
        int numVertices = graph->numberOfVertex();
        int numEdges = graph->numberOfEdges() * 2;      

		// create device event timers
		hipEventCreate(&start);
		hipEventCreate(&end);

        // allocate the memory for the arrays
        int * vertexArray = new int [numVertices + 1];
        int * edgeArray = new int [numEdges];


        // start GPU timer
        hipEventRecord( start, 0 );

        // convert the graph to GPU format (one dim array)
        convertAdjListToArray(graph, vertexArray, edgeArray);

        // create a thread for each GPU
        std::thread threadList[numDevices];

		// 
		int i = 0;
		while( i < numDevices )
		{

				// launch one thread
				threadList[i] = std::thread(multipleGPUThreadFunction, vertexArray, edgeArray, numVertices, numEdges, source, i, gpuCost, numIngress, numDevices);

				// increment number of devices
				i++;
		
        }

        // wait for any remaining threads
        for(int j = 0; j < numDevices; j++)
		{
			threadList[j].join();
		}

        // stop the GPU timer
        hipEventRecord( end, 0 );
        hipEventSynchronize( end );

        // calculate the amount of execution time
        float deviceWithTransferTime;
        hipEventElapsedTime( &deviceWithTransferTime, start, end );

        // clean up timer variables
        hipEventDestroy(start);
    	hipEventDestroy(end);

    	// clean up all the CPU memory
    	delete [] vertexArray;
    	delete [] edgeArray;


        // return the execution time
   	    return (deviceWithTransferTime /1000.0);
   	    
    }


void multipleGPUThreadFunction(int * vertexArray, int * edgeArray, int numVertices, int numEdges, int * source, int deviceNum, int ** gpuCost, int numIngress, int numDevices)
{
        // declare variables
        int numBlocks, numThreads;
        bool searching = true;

        // set the device number
        hipSetDevice(deviceNum); 	

        for(int n = deviceNum; n < numIngress; n += numDevices )
        {
            // allocate memory for the arrays on the Host
            int * costArray = new int [numVertices];
            bool * frontierArray = new bool [numVertices];
            bool * frontierUpdatingArray = new bool [numVertices];
            bool * visitedArray = new bool [numVertices];

            // initialize array values
            for(int i = 0; i < numVertices; i++)
            {
                costArray[i] = -1;
                frontierArray[i] = false;
                frontierUpdatingArray[i] = false;
                visitedArray[i] = false;
            }

            // set values for source node
            frontierArray[source[n]] = true;
            visitedArray[source[n]] = true;
            costArray[source[n]] = 0;


            // create the cuda memory and send to the device 

                // vertex array
                int* deviceVertexArray;
                hipMalloc( (void**) &deviceVertexArray, sizeof(int)*(numVertices + 1));
                hipMemcpy( deviceVertexArray, vertexArray, sizeof(int)*(numVertices + 1), hipMemcpyHostToDevice);


                // edge array
                int* deviceEdgeArray;
                hipMalloc( (void**) &deviceEdgeArray, sizeof(int)*numEdges);
                hipMemcpy( deviceEdgeArray, edgeArray, sizeof(int)*numEdges, hipMemcpyHostToDevice);

                // cost array
                int* deviceCostArray;
                hipMalloc( (void**) &deviceCostArray, sizeof(int)*numVertices);
                hipMemcpy( deviceCostArray, costArray, sizeof(int)*numVertices, hipMemcpyHostToDevice);

                // frontier array
                bool* deviceFrontierArray;
                hipMalloc( (void**) &deviceFrontierArray, sizeof(bool)*numVertices);
                hipMemcpy( deviceFrontierArray, frontierArray, sizeof(bool)*numVertices, hipMemcpyHostToDevice);

                // frontier updating array
                bool* deviceFrontierUpdatingArray;
                hipMalloc( (void**) &deviceFrontierUpdatingArray, sizeof(bool)*numVertices);
                hipMemcpy( deviceFrontierUpdatingArray, frontierUpdatingArray, sizeof(bool)*numVertices, hipMemcpyHostToDevice);

                // visited array
                bool* deviceVisitedArray;
                hipMalloc( (void**) &deviceVisitedArray, sizeof(bool)*numVertices);
                hipMemcpy( deviceVisitedArray, visitedArray, sizeof(bool)*numVertices, hipMemcpyHostToDevice);

                // bool to stop BFS
                bool * deviceSearching;
                hipMalloc( (void**) &deviceSearching, sizeof(bool));


            // calculate the number of blocks and threads
            if(numVertices > MAX_THREADS_PER_BLOCK)
            {
                numBlocks = (int) ceil(numVertices/(double)MAX_THREADS_PER_BLOCK); 
                numThreads = MAX_THREADS_PER_BLOCK; 
            }
            else
            {
                numBlocks = 1;
                numThreads = numVertices;
            }

            // start BFS search
            while(searching)
            {
            	// set the searching value to false (stop the search)
                searching = false;

                // copy the search value to the GPU
                hipMemcpy(deviceSearching, &searching, sizeof(bool), hipMemcpyHostToDevice);

                // start BFS kernel
                bfs_kernel<<<numBlocks, numThreads>>>(deviceVertexArray, deviceEdgeArray, deviceFrontierArray, deviceFrontierUpdatingArray, deviceVisitedArray, deviceCostArray, numVertices);

              	// start bfs update array kernel
                bfsUpdateArrays_kernel<<<numBlocks, numThreads>>>(deviceFrontierArray, deviceFrontierUpdatingArray, deviceVisitedArray, deviceSearching, numVertices);

                // get the search value back from the GPU
                hipMemcpy(&searching, deviceSearching, sizeof(bool), hipMemcpyDeviceToHost);
            }

            // copy the cost array from the GPU
            hipMemcpy( gpuCost[n], deviceCostArray, sizeof(int)*numVertices, hipMemcpyDeviceToHost);

        	// clean up all the CPU memory
        	delete [] costArray;
        	delete [] frontierArray;
        	delete [] frontierUpdatingArray;
        	delete [] visitedArray;

        	// clean up all the CUDA memory
            hipFree(deviceVertexArray);
            hipFree(deviceEdgeArray);
            hipFree(deviceCostArray);
            hipFree(deviceFrontierArray);
            hipFree(deviceFrontierUpdatingArray);
            hipFree(deviceVisitedArray);	
        }
}        



void convertAdjListToArray(Graph * G, int * vertexArray, int * edgeArray)
    {
        // declare variables
        int edgeCount = 0;

        // loop through all the vertices
        for (int i = 0; i < G->graphVector.size(); i++)
        {
            vertexArray[i] = edgeCount;

            for (auto it : G->graphVector[i].edges)
            {
                // put the destination in the edge array
                edgeArray[edgeCount] = std::get<2>(it).get_Vertex_ID();
                
                // increment the edge count
                edgeCount ++;
            }   
        }

        // add the index of the last edge (makes it easier to loop)
        vertexArray[G->graphVector.size()] = edgeCount;
    }      
           


#endif
